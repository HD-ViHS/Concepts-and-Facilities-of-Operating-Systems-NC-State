// Elapsed Real Time for input-5.txt: 0m0.963s
// GPU for this execution time: rtx2060

#include <stdio.h>
#include <stdbool.h>
#include <hip/hip_runtime.h>

// Input sequence of values.
int *vList;

// Number of values on the list.
int vCount = 0;

// Capacity of the list of values.
int vCap = 0;

// General function to report a failure and exit.
static void fail( char const *message ) {
  fprintf( stderr, "%s\n", message );
  exit( 1 );
}

// Print out a usage message, then exit.
static void usage() {
  printf( "usage: zerosum [report]\n" );
  exit( 1 );
}

// Read the list of values.
__host__ void readList() {
  // Set up initial list and capacity.
  vCap = 5;
  vList = (int *) malloc( vCap * sizeof( int ) );

  // Keep reading as many values as we can.
  int v;
  while ( scanf( "%d", &v ) == 1 ) {
    // Grow the list if needed.
    if ( vCount >= vCap ) {
      vCap *= 2;
      vList = (int *) realloc( vList, vCap * sizeof( int ) );
    }

    // Store the latest value in the next array slot.
    vList[ vCount++ ] = v;
  }
}

__global__ void checkSum( int vCount, bool report, int* input, int* output ) {
  int idx = blockDim.x * blockIdx.x + threadIdx.x;

  if ( idx < vCount ) {
    int total = 0;
    for(int i = idx; i < vCount; i++) {
      total += input[i];
      if(total == 0) {
	output[idx]++;
	if(report)
	  printf("%d .. %d\n", idx, i);
      }
    }
  }
}

int main( int argc, char *argv[] ) {
  if ( argc < 1 || argc > 2 )
    usage();

  // If there's an argument, it better be "report"
  bool report = false;
  if ( argc == 2 ) {
    if ( strcmp( argv[ 1 ], "report" ) != 0 )
      usage();
    report = true;
  }

  readList();

  // Allocate space on the device to hold a copy of the sequence.

  int *vListCopy = NULL;
  hipMalloc( (void **)&vListCopy, vCount * sizeof( int ) );

  // Copy the sequence over to the device.

  hipMemcpy( vListCopy, vList, vCount * sizeof( int ), hipMemcpyHostToDevice );

  // Maybe some more code written by you.

  int *countListDevice = NULL;
  hipMalloc( (void **)&countListDevice, vCount * sizeof( int ) );

  // Block and grid dimensions.
  int threadsPerBlock = 100;
  // Round up for the number of blocks we need.
  int blocksPerGrid = ( vCount + threadsPerBlock - 1 ) / threadsPerBlock;

  // Run our kernel on these block/grid dimensions
  checkSum<<<blocksPerGrid, threadsPerBlock>>>( vCount, report, vListCopy, countListDevice );
  if ( hipGetLastError() != hipSuccess )
    fail( "Failure in CUDA kernel execution." );

  // Add code to copy results back to the host and then add up the total
  // number of zero-sum ranges found.

  int *countListHost = (int *)malloc(vCount * sizeof( int ));

  hipMemcpy( countListHost, countListDevice, vCount * sizeof( int ), hipMemcpyDeviceToHost );
 
  int grandTotal = 0;
  for(int i = 0; i < vCount; i++) {
    grandTotal += countListHost[i];
  }

  printf("Total: %d\n", grandTotal);

  // Free memory on the device and the host.
  hipFree(countListDevice);
  hipFree(vListCopy);
  free(countListHost);
  free( vList );

  hipDeviceReset();

  return 0;
}
