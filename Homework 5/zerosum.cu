// Elapsed Real Time for input-5.txt:
// GPU for this execution time:

#include <stdio.h>
#include <stdbool.h>
#include <hip/hip_runtime.h>

// Input sequence of values.
int *vList;

// Number of values on the list.
int vCount = 0;

// Capacity of the list of values.
int vCap = 0;

// General function to report a failure and exit.
static void fail( char const *message ) {
  fprintf( stderr, "%s\n", message );
  exit( 1 );
}

// Print out a usage message, then exit.
static void usage() {
  printf( "usage: zerosum [report]\n" );
  exit( 1 );
}

// Read the list of values.
__host__ void readList() {
  // Set up initial list and capacity.
  vCap = 5;
  vList = (int *) malloc( vCap * sizeof( int ) );

  // Keep reading as many values as we can.
  int v;
  while ( scanf( "%d", &v ) == 1 ) {
    // Grow the list if needed.
    if ( vCount >= vCap ) {
      vCap *= 2;
      vList = (int *) realloc( vList, vCap * sizeof( int ) );
    }

    // Store the latest value in the next array slot.
    vList[ vCount++ ] = v;
  }
}

__global__ void checkSum( int vCount, bool report ) {
  int idx = blockDim.x * blockIdx.x + threadIdx.x;

  if ( idx < vCount ) {
    // ...
  }
}

int main( int argc, char *argv[] ) {
  if ( argc < 1 || argc > 2 )
    usage();

  // If there's an argument, it better be "report"
  bool report = false;
  if ( argc == 2 ) {
    if ( strcmp( argv[ 1 ], "report" ) != 0 )
      usage();
    report = true;
  }

  readList();

  // Allocate space on the device to hold a copy of the sequence.

  // Copy the sequence over to the device.

  // Maybe some more code written by you.

  // Block and grid dimensions.
  int threadsPerBlock = 100;
  // Round up for the number of blocks we need.
  int blocksPerGrid = ( vCount + threadsPerBlock - 1 ) / threadsPerBlock;

  // Run our kernel on these block/grid dimensions
  checkSum<<<blocksPerGrid, threadsPerBlock>>>( vCount, report );
  if ( hipGetLastError() != hipSuccess )
    fail( "Failure in CUDA kernel execution." );

  // Add code to copy results back to the host and then add up the total
  // number of zero-sum ranges found.

  // Free memory on the device and the host.
  // ..
  free( vList );

  hipDeviceReset();

  return 0;
}
